#include <hip/hip_runtime.h>

#include <iostream>
#include <vector>
#include <string>
#include <chrono>
#include <random>

using namespace std::chrono;

static const int G_INIT   = 1;
static const int G_EXT    = 1;
static const int MATCH    = 1;
static const int MISMATCH = -1;

__device__ __forceinline__
int device_score(unsigned char a, unsigned char b) {
    return (a == b) ? MATCH : MISMATCH;
}

__global__
void sw_kernel_diag(
    int d,
    int n,
    int m,
    const unsigned char* __restrict__ seq1,
    const unsigned char* __restrict__ seq2,
    int* __restrict__ H,
    int* __restrict__ E,
    int* __restrict__ F
) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int start_i = max(1, d - n);
    int end_i   = min(d - 1, m);
    int diag_len = end_i - start_i + 1;

    if (tid >= diag_len) return;

    int i = start_i + tid;
    int j = d - i;

    int idx      = i * (n + 1) + j;
    int idx_up   = (i - 1) * (n + 1) + j;
    int idx_left = i * (n + 1) + (j - 1);
    int idx_diag = (i - 1) * (n + 1) + (j - 1);

    int e = max(H[idx_left] - G_INIT, E[idx_left] - G_EXT);
    int f = max(H[idx_up]   - G_INIT, F[idx_up]   - G_EXT);
    int match = H[idx_diag] + device_score(seq1[j - 1], seq2[i - 1]);

    int h_val = max(0, max(match, max(e, f)));

    E[idx] = e;
    F[idx] = f;
    H[idx] = h_val;
}

extern "C" int SmithWatermanLazyGPU(const unsigned char* seq1, const unsigned char* seq2, int n, int m) {
    size_t matrix_size = (size_t)(m + 1) * (size_t)(n + 1);

    int *dH = nullptr, *dE = nullptr, *dF = nullptr;
    hipMalloc(&dH, matrix_size * sizeof(int));
    hipMalloc(&dE, matrix_size * sizeof(int));
    hipMalloc(&dF, matrix_size * sizeof(int));
    hipMemset(dH, 0, matrix_size * sizeof(int));
    hipMemset(dE, 0, matrix_size * sizeof(int));
    hipMemset(dF, 0, matrix_size * sizeof(int));

    unsigned char *d_seq1 = nullptr, *d_seq2 = nullptr;
    hipMalloc(&d_seq1, n * sizeof(unsigned char));
    hipMalloc(&d_seq2, m * sizeof(unsigned char));
    hipMemcpy(d_seq1, seq1, n * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(d_seq2, seq2, m * sizeof(unsigned char), hipMemcpyHostToDevice);

    const int THREADS = 256;
    for (int d = 2; d <= n + m; ++d) {
        int start_i = std::max(1, d - n);
        int end_i   = std::min(d - 1, m);
        int diag_len = (end_i >= start_i) ? (end_i - start_i + 1) : 0;
        if (diag_len <= 0) continue;
        int blocks = (diag_len + THREADS - 1) / THREADS;
        sw_kernel_diag<<<blocks, THREADS>>>(d, n, m, d_seq1, d_seq2, dH, dE, dF);
        hipDeviceSynchronize();
    }

    std::vector<int> h_H(matrix_size);
    hipMemcpy(h_H.data(), dH, matrix_size * sizeof(int), hipMemcpyDeviceToHost);

    int best = 0;
    for (int val : h_H) best = std::max(best, val);

    hipFree(d_seq1);
    hipFree(d_seq2);
    hipFree(dH);
    hipFree(dE);
    hipFree(dF);

    return best;
}

void runLazy() {
    std::mt19937_64 rng(42);
    std::uniform_int_distribution<int> dist(0, 3);
    const char* nts = "ACGT";

    std::vector<std::pair<std::string, std::string>> tests = {
        {"AABDADB", "AADCBAB"},
        {"AAA", "AAA"},
        {std::string(5000, 'A'), std::string(5000, 'A')},
        {std::string(2000, 'A'), std::string(2000, 'T')},
    };

    for (int i = 0; i < 5; ++i) {
        std::string s1, s2;
        for (int j = 0; j < 4000 + i * 1000; ++j) {
            s1 += nts[dist(rng)];
            s2 += nts[dist(rng)];
        }
        tests.emplace_back(std::move(s1), std::move(s2));
    }

    int correct = 0;
    for (size_t i = 0; i < tests.size(); ++i) {
        const auto& tc = tests[i];
        int n = (int)tc.first.size();
        int m = (int)tc.second.size();

        std::vector<unsigned char> h_seq1(n), h_seq2(m);
        for (int j = 0; j < n; ++j) h_seq1[j] = (unsigned char)tc.first[j];
        for (int j = 0; j < m; ++j) h_seq2[j] = (unsigned char)tc.second[j];

        int lazy = SmithWatermanLazyGPU(h_seq1.data(), h_seq2.data(), n, m);

        std::cout << "Test " << i+1 << " (" << n << "x" << m << ")\n";
        std::cout << "  Lazy GPU  = " << lazy << "\n";
    }

}
 

// int main() {
//     runLazy();
//     return 0;
// }